#include "hip/hip_runtime.h"
#include <fftw3.h>
#include <complex>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/universal_vector.h>
#include <vector>
#include <random>
#include <algorithm>
#include <iterator>
#include <iostream>
#include <hipfft/hipfft.h>
#include "cuda_safe_call.h"
#include "cufft_safe_call.h"
#include <chrono>
#include <execution>
#include <fstream>
#include <string>
#include <ios>
#include <unistd.h>


template<typename T>
T norm(const std::vector<T>& data)
{
    T sum{0};
    for(const auto &x: data)
    {
        sum += x*x;
    }

    return std::sqrt(sum);
}

template<typename T>
T norm(const std::vector< std::complex<T> >& data)
{
    T sum{0};
    for(const auto &x: data)
    {
        sum += real(x*conj(x));
    }

    return std::sqrt(sum);
}

template <class Vec>
void plot_vec(const Vec& data)
{
    for(auto x: data)
    {
        std::cout << x << " ";
    }
    std::cout << std::endl;
}

template <class VecDev, class VecHost>
void device_to_std_vec(const VecDev& vec_d, VecHost& vec_h)
{
    using T = typename VecHost::value_type;
    std::size_t n_el = vec_h.size();
    std::size_t count = n_el*sizeof(T);
    auto host_ref = vec_h.data();
    auto dev_ref = thrust::raw_pointer_cast( vec_d.data() );
    CUDA_SAFE_CALL( hipMemcpy( (void*) host_ref, (const void*) dev_ref, count, hipMemcpyDeviceToHost ) );
    
}


template <class VecDev>
std::vector< typename VecDev::value_type > device_to_std_vec(const VecDev& vec_d)
{
    using T = typename VecDev::value_type;
    std::size_t n_el = vec_d.size();
    std::size_t count = n_el*sizeof(T);
    std::vector< T > vec_h(n_el);
    auto host_ref = vec_h.data();
    auto dev_ref = thrust::raw_pointer_cast( vec_d.data() );
    CUDA_SAFE_CALL( hipMemcpy( (void*) host_ref, (const void*) dev_ref, count, hipMemcpyDeviceToHost ) );
    return vec_h; 
}


class memory_info
{
public:
    memory_info()
    {}
    ~memory_info(){}

    void print_mem(const std::string& mem_ = "")
    {
        get_current_memory();
        std::cout << "MEM " << mem_ << ": host = " << host_occupied_mem_in_kB << "kB, device = " << device_occupied_mem_in_kB << "kB, total = " << host_occupied_mem_in_kB+device_occupied_mem_in_kB << "kB." << std::endl;
    }


private:
    std::string pid, comm, state, ppid, pgrp, session, tty_nr;
    std::string tpgid, flags, minflt, cminflt, majflt, cmajflt;
    std::string utime, stime, cutime, cstime, priority, nice;
    std::string O, itrealvalue, starttime, vsize;
    hipDeviceProp_t device_prop;
    std::size_t rss;
    std::size_t host_occupied_mem_in_kB;
    std::size_t device_free_mem, device_total_mem, device_occupied_mem_in_kB;
    std::size_t page_size_kb = sysconf(_SC_PAGE_SIZE)/1024; //in kB, usually 2MB per page

    void get_current_memory()
    {
        std::ifstream stat_stream("/proc/self/stat", std::ios_base::in);
        stat_stream >> pid >> comm >> state >> ppid >> pgrp >> session >> tty_nr
           >> tpgid >> flags >> minflt >> cminflt >> majflt >> cmajflt
           >> utime >> stime >> cutime >> cstime >> priority >> nice
           >> O >> itrealvalue >> starttime >> vsize >> rss;
        stat_stream.close();
        CUDA_SAFE_CALL( hipMemGetInfo ( &device_free_mem, &device_total_mem ) ); 
        CUDA_SAFE_CALL( hipDeviceSynchronize() );
        host_occupied_mem_in_kB = rss*page_size_kb;
        device_occupied_mem_in_kB = (device_total_mem - device_free_mem)/1024;

    }

};


template<class T>
class cuda_universal_vector
{
public:
    cuda_universal_vector(){};
    cuda_universal_vector(std::size_t size):
    size_(size)
    {
        CUDA_SAFE_CALL( hipMallocManaged((void**)&data, sizeof(T)*size_) );
    }
    ~cuda_universal_vector()
    {
        if(data != nullptr)
        {
            hipFree(data);
        }
    }
    void init(std::size_t size)
    {
        if(size_ == 0)
        {
            size_ = size;
            CUDA_SAFE_CALL( hipMallocManaged((void**)&data, sizeof(T)*size_) );
        }
    }
    T* raw_ptr()
    {
        return data;
    }
    void copy_to_this_vector(const T* other)
    {
        CUDA_SAFE_CALL( hipMemcpy(data, other, sizeof(T)*size_, hipMemcpyHostToDevice ) );
    }
    void copy_from_this_vector(T* other)
    {
        CUDA_SAFE_CALL( hipMemcpy(other, data, sizeof(T)*size_, hipMemcpyDeviceToHost ) );
    }

private:
    T* data = nullptr;
    std::size_t size_ = 0;

};


template<typename T>
struct select_cufft_type
{
};
template<>
struct select_cufft_type<float>
{
    using real = hipfftReal;
    using complex = hipfftComplex;
};
template<>
struct select_cufft_type<double>
{
    using real = hipfftDoubleReal;
    using complex = hipfftDoubleComplex;
};

int main(int argc, char const *argv[])
{
    using T = double;
    using TRealCufft = select_cufft_type<T>::real;
    using TComplexCufft = select_cufft_type<T>::complex;

    if(argc != 5)
    {
        std::cout << "usage: " << argv[0] << " N CPU GPU_number use_thrust_universal_vector" << std::endl;
        std::cout << "  where N is the size of a cube," << std::endl;
        std::cout << "  CPU=y/n is the use of CPU fftw for verification or not." << std::endl;
        std::cout << "  use_thrust_universal_vector=y/n is the use of thrust::universal_vector ('y') or hipMallocManaged ('n')." << std::endl;
        return 0;
    }
    int device_id = std::stoi(argv[3]);
    hipDeviceProp_t device_prop;
    CUDA_SAFE_CALL( hipGetDeviceProperties(&device_prop, device_id) );     
    std::cout << "using CUDA device number " << device_id << ": " << device_prop.name << std::endl;
    CUDA_SAFE_CALL( hipSetDevice(device_id) );

    std::size_t N_size = std::stoi(argv[1]);
    char use_fftw = argv[2][0];
    char use_thrust_for_cuda_data = argv[4][0];
    std::size_t N = N_size, M = N_size, L = N_size;
    std::size_t L_reduced = L/2+1; 

    std::vector< std::complex<T> > data_c(N*M*L_reduced);
    std::vector< T > data_r_1(N*M*L);
    std::vector< T > data_r_2(N*M*L);

    memory_info mem;

    std::cout << "initializing vector of randoms" << std::endl;
    mem.print_mem();

    {
        std::random_device rd;
        std::mt19937 engine{ rd() }; 
        std::uniform_real_distribution<> dist(-100.0, 100.0);

        auto gen_rand = [&dist, &engine]()
        {
            return dist(engine);
        };        
        std::generate(std::execution::par, begin(data_r_1), end(data_r_1), gen_rand);
    }
    //timers
 
    //FFTW part

        
    std::cout << "executing fftw...";
    mem.print_mem();
    std::cout << std::flush;

    fftw_complex* c_fftw = (fftw_complex*)( data_c.data() );
    T* r_fftw = data_r_1.data();
    T* r2_fftw = data_r_2.data();

    fftw_plan plan_fftw_r2c, plan_fftw_c2r;
    auto start_0 = std::chrono::high_resolution_clock::now();
    if(use_fftw == 'y')
    {
        plan_fftw_r2c = fftw_plan_dft_r2c_3d(N, M, L, r_fftw, c_fftw, FFTW_ESTIMATE);
        plan_fftw_c2r = fftw_plan_dft_c2r_3d(N, M, L, c_fftw, r2_fftw, FFTW_ESTIMATE);
    
        fftw_execute(plan_fftw_r2c);
        fftw_execute(plan_fftw_c2r);
        fftw_destroy_plan(plan_fftw_r2c);
        fftw_destroy_plan(plan_fftw_c2r);
    }
    auto stop_0 = std::chrono::high_resolution_clock::now();


    if(use_fftw == 'y')
        std::transform(data_r_2.cbegin(), data_r_2.cend(), data_r_2.begin(), [&N, &M, &L]( T c) { return c/(N*M*L); });

    
    std::cout << "done." << std::endl;
    //CUDA part
    std::cout << "executing cufft...";
    std::cout << std::flush;

    

    hipfftHandle cufft_handle_r2c, cufft_handle_c2r;

    mem.print_mem("starting cuda");
    hipEvent_t start_1, stop_1;
    CUDA_SAFE_CALL(hipEventCreate(&start_1));
    CUDA_SAFE_CALL(hipEventCreate(&stop_1));    
    CUDA_SAFE_CALL(hipEventRecord(start_1));

    CUFFT_SAFE_CALL(hipfftCreate(&cufft_handle_r2c));
    CUFFT_SAFE_CALL(hipfftCreate(&cufft_handle_c2r));
    CUDA_SAFE_CALL(hipDeviceSynchronize());


// We ask cuFFT to not allocate any buffers automatically
    CUFFT_SAFE_CALL(hipfftSetAutoAllocation(cufft_handle_r2c, false));
    CUFFT_SAFE_CALL(hipfftSetAutoAllocation(cufft_handle_c2r, false));
    CUDA_SAFE_CALL(hipDeviceSynchronize());

// estimate buffer sizes
    std::size_t scratch_sizes[2];
    CUFFT_SAFE_CALL(hipfftMakePlan3d(cufft_handle_r2c, N, M, L, HIPFFT_D2Z, &scratch_sizes[0]));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUFFT_SAFE_CALL(hipfftMakePlan3d(cufft_handle_c2r, N, M, L, HIPFFT_Z2D, &scratch_sizes[1]));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    double to_gb = 1.0/(1024.0*1024.0*1024.0);
    std::cout << "D2Z buffer size = " << scratch_sizes[0]*to_gb << "GB, Z2D buffer size = " << scratch_sizes[1]*to_gb << "GB." << std::endl;
    std::size_t bufer_size = scratch_sizes[0]>scratch_sizes[1]?scratch_sizes[0]:scratch_sizes[1];

// allocating buffer size
    thrust::universal_vector< T > buffer_dev_1;
    cuda_universal_vector< T > buffer_dev_2;

    if(use_thrust_for_cuda_data == 'y')
    {
        buffer_dev_1 = thrust::universal_vector< T >(bufer_size);
        auto buffer_dev_c = thrust::raw_pointer_cast( buffer_dev_1.data() );

        CUFFT_SAFE_CALL(hipfftSetWorkArea(cufft_handle_r2c, buffer_dev_c));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        CUFFT_SAFE_CALL(hipfftSetWorkArea(cufft_handle_c2r, buffer_dev_c));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        std::cout << "buffer for cufft allocated" << std::endl;        
    }
    else
    {
        buffer_dev_2.init(bufer_size);
        auto buffer_dev_c = buffer_dev_2.raw_ptr();

        CUFFT_SAFE_CALL(hipfftSetWorkArea(cufft_handle_r2c, buffer_dev_c));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        CUFFT_SAFE_CALL(hipfftSetWorkArea(cufft_handle_c2r, buffer_dev_c));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        std::cout << "buffer for cufft allocated" << std::endl;          
    }
    

    mem.print_mem();

    if(use_thrust_for_cuda_data == 'y')
    {
        thrust::universal_vector< T > data_r_1_dev(data_r_1); //input vector copy to device
        thrust::universal_vector< thrust::complex<T> > data_c_dev(N*M*L_reduced);
        thrust::universal_vector< T > data_r_2_dev(data_r_2);
        auto data_c_dev_c = thrust::raw_pointer_cast( data_c_dev.data() );
        auto data_r_1_dev_c = thrust::raw_pointer_cast( data_r_1_dev.data() );
        auto data_r_2_dev_c = thrust::raw_pointer_cast( data_r_2_dev.data() );
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        mem.print_mem("thrust device vectors allocated");

        
        for(int j=0;j<100;j++)
        {
            std::cout << "D2Z execution" << std::endl;
            CUFFT_SAFE_CALL( hipfftExecD2Z(cufft_handle_r2c, static_cast<TRealCufft*>( data_r_1_dev_c ), (TComplexCufft*)( data_c_dev_c ) ) ); //only works with C-style cast!
            CUDA_SAFE_CALL(hipDeviceSynchronize());
            std::cout << "Z2D execution" << std::endl;
            CUFFT_SAFE_CALL( hipfftExecZ2D(cufft_handle_c2r, (TComplexCufft*)( data_c_dev_c ), static_cast<TRealCufft*>( data_r_2_dev_c ) ) ); //only works with C-style cast!
            CUDA_SAFE_CALL(hipDeviceSynchronize());
            mem.print_mem("thrust it: " + std::to_string(j) );
        }
    }
    else
    {
        cuda_universal_vector< T > data_r_1_dev(N*M*L); //input vector copy to device
        data_r_1_dev.copy_to_this_vector( data_r_1.data() );

        cuda_universal_vector< TComplexCufft > data_c_dev(N*M*L_reduced);

        cuda_universal_vector< T > data_r_2_dev(N*M*L);
        data_r_2_dev.copy_to_this_vector( data_r_2.data() );

        auto data_c_dev_c = data_c_dev.raw_ptr() ;
        auto data_r_1_dev_c = data_r_1_dev.raw_ptr();
        auto data_r_2_dev_c = data_r_2_dev.raw_ptr();
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        mem.print_mem("cuda device vectors allocated");

        
        for(int j=0;j<100;j++)
        {
            std::cout << "D2Z execution" << std::endl;
            CUFFT_SAFE_CALL( hipfftExecD2Z(cufft_handle_r2c, data_r_1_dev_c, data_c_dev_c  ) ); //only works with C-style cast!
            CUDA_SAFE_CALL(hipDeviceSynchronize());
            std::cout << "Z2D execution" << std::endl;
            CUFFT_SAFE_CALL( hipfftExecZ2D(cufft_handle_c2r, data_c_dev_c , data_r_2_dev_c ) ); //only works with C-style cast!
            CUDA_SAFE_CALL(hipDeviceSynchronize());
            mem.print_mem("cuda it: " + std::to_string(j) );
        }        
    }

    CUFFT_SAFE_CALL( hipfftDestroy(cufft_handle_r2c) );
    CUFFT_SAFE_CALL( hipfftDestroy(cufft_handle_c2r) );
    CUDA_SAFE_CALL(hipDeviceSynchronize());


    CUDA_SAFE_CALL(hipEventRecord(stop_1));
    CUDA_SAFE_CALL(hipEventSynchronize(stop_1));
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    std::cout << "cufft done." << std::endl;
    mem.print_mem();

    float duration_1 = 0;
    CUDA_SAFE_CALL(hipEventElapsedTime(&duration_1, start_1, stop_1));
    auto duration_0 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_0 - start_0);
    std::cout << " fftw time = " << duration_0.count() << " cufft time = " << duration_1 << std::endl;
    

    if((use_fftw == 'y')&&(use_thrust_for_cuda_data == 'y'))
    {
        //thrust::host_vector< T > data_r_2_host(data_r_2_dev);
        //check diff
        // thrust::host_vector< thrust::complex<T> > data_c_host(data_c_dev);        
        // auto data_r_2_host = device_to_std_vec(data_r_2_dev);

        // std::vector<T> data_r_2_from_cuda(N*M*L);

        // device_to_std_vec(data_r_2_dev, data_r_2_from_cuda);


        // //plot_vec(data_c);
        // //std::cout << "..." << std::endl;
        // //plot_vec(data_c_host);


        // std::transform(data_r_2_host.cbegin(), data_r_2_host.cend(), data_r_2_host.begin(), [&N, &M, &L]( T c) { return c/(N*M*L); });  


        // std::vector< std::complex<T> > diff_c_cufft_vs_fftw(N*M*L_reduced);
        // std::transform(data_c.begin(), data_c.end(), data_c_host.begin(), diff_c_cufft_vs_fftw.begin(), [](auto c, auto d){ return static_cast<std::complex<T> >(c) - static_cast<std::complex<T> >(d); } );

        // auto c_diff = norm(diff_c_cufft_vs_fftw);  
        // std::cout << "cufft vs fftw complex difference: " << c_diff << std::endl;
        // if (c_diff/(N*M*L)>1.0e-10)
        // {
        //     std::cout << "fftw(u(0)) = " << data_c[0] << " cufft(u(0)) = " << data_c_host[0] << std::endl;
        // }



        // std::vector<T> diff_r_fftw(N*M*L);
        // std::transform(data_r_1.begin(), data_r_1.end(), data_r_2.begin(), diff_r_fftw.begin(), std::minus< T >() );
        // std::vector<T> diff_r_cufft(N*M*L);
        // std::transform(data_r_1.begin(), data_r_1.end(), data_r_2_host.begin(), diff_r_cufft.begin(), std::minus< T >() );

        // std::vector<T> diff_r_fftw_vs_cufft(N*M*L);
        //  std::transform(data_r_2.begin(), data_r_2.end(), data_r_2_host.begin(), diff_r_fftw_vs_cufft.begin(), std::minus< T >() );   
        

        // std::cout << "fftw complex vector norm: " << norm(data_c) << std::endl;

        // std::cout << "fftw result difference: " << norm(diff_r_fftw) << std::endl;
        // std::cout << "cufft result difference: " << norm(diff_r_cufft) << std::endl;
        // std::cout << "cufft vs fftw result difference: " << norm(diff_r_fftw_vs_cufft) << std::endl;

    }

    return 0;

}


