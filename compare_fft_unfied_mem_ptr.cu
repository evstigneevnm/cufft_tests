#include "hip/hip_runtime.h"
#include <fftw3.h>
#include <complex>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/universal_vector.h>
#include <vector>
#include <random>
#include <algorithm>
#include <iterator>
#include <iostream>
#include <hipfft/hipfft.h>
#include "cuda_safe_call.h"
#include "cufft_safe_call.h"
#include <chrono>

template<typename T>
T norm(const std::vector<T>& data)
{
    T sum{0};
    for(const auto &x: data)
    {
        sum += x*x;
    }

    return std::sqrt(sum);
}

template<typename T>
T norm(const std::vector< std::complex<T> >& data)
{
    T sum{0};
    for(const auto &x: data)
    {
        sum += real(x*conj(x));
    }

    return std::sqrt(sum);
}

template <class Vec>
void plot_vec(const Vec& data)
{
    for(auto x: data)
    {
        std::cout << x << " ";
    }
    std::cout << std::endl;
}

template <class VecDev, class VecHost>
void device_to_std_vec(const VecDev& vec_d, VecHost& vec_h)
{
    using T = typename VecHost::value_type;
    std::size_t n_el = vec_h.size();
    std::size_t count = n_el*sizeof(T);
    auto host_ref = vec_h.data();
    auto dev_ref = thrust::raw_pointer_cast( vec_d.data() );
    CUDA_SAFE_CALL( hipMemcpy( (void*) host_ref, (const void*) dev_ref, count, hipMemcpyDeviceToHost ) );
    
}


template <class VecDev>
std::vector< typename VecDev::value_type > device_to_std_vec(const VecDev& vec_d)
{
    using T = typename VecDev::value_type;
    std::size_t n_el = vec_d.size();
    std::size_t count = n_el*sizeof(T);
    std::vector< T > vec_h(n_el);
    auto host_ref = vec_h.data();
    auto dev_ref = thrust::raw_pointer_cast( vec_d.data() );
    CUDA_SAFE_CALL( hipMemcpy( (void*) host_ref, (const void*) dev_ref, count, hipMemcpyDeviceToHost ) );
    return vec_h; 
}

template<typename T>
struct select_cufft_type
{
};
template<>
struct select_cufft_type<float>
{
    using real = hipfftReal;
    using complex = hipfftComplex;
};
template<>
struct select_cufft_type<double>
{
    using real = hipfftDoubleReal;
    using complex = hipfftDoubleComplex;
};



int main(int argc, char const *argv[])
{
    using T = double;
    using TRealCufft = select_cufft_type<T>::real;
    using TComplexCufft = select_cufft_type<T>::complex;

    if(argc != 4)
    {
        std::cout << "usage: " << argv[0] << " N CPU GPU_number" << std::endl;
        std::cout << "  where N is the size of a cube," << std::endl;
        std::cout << "  CPU=y/n is use CPU fftw for verification or not." << std::endl;
        return 0;
    }
    int device_id = std::atoi(argv[3]);
    hipDeviceProp_t device_prop;
    CUDA_SAFE_CALL( hipGetDeviceProperties(&device_prop, device_id) ); 	
    std::cout << "using CUDA device number " << device_id << ": " << device_prop.name << std::endl;
    CUDA_SAFE_CALL( hipSetDevice(device_id) );

    std::size_t N_size = std::atoi(argv[1]);
    char use_fftw = argv[2][0];
    std::size_t N = N_size, M = N_size, L = N_size;
    std::size_t L_reduced = L/2+1; 

    std::vector< std::complex<T> > data_c(N*M*L_reduced);
    std::vector< T > data_r_1(N*M*L);
    std::vector< T > data_r_2(N*M*L);

    std::random_device rd;
    std::mt19937 engine{ rd() }; 
    std::uniform_real_distribution<> dist(0.0, 100.0);

    auto gen_rand = [&dist, &engine]()
    {
        return dist(engine);
    };
    std::generate(begin(data_r_1), end(data_r_1), gen_rand);
    
    //timers
 
    //FFTW part

    
    std::cout << "executing fftw...";
    std::cout << std::flush;

    fftw_complex* c_fftw = (fftw_complex*)( data_c.data() );
    T* r_fftw = data_r_1.data();
    T* r2_fftw = data_r_2.data();

    fftw_plan plan_fftw_r2c, plan_fftw_c2r;
    auto start_0 = std::chrono::high_resolution_clock::now();
    if(use_fftw == 'y')
    {
        plan_fftw_r2c = fftw_plan_dft_r2c_3d(N, M, L, r_fftw, c_fftw, FFTW_ESTIMATE);
        plan_fftw_c2r = fftw_plan_dft_c2r_3d(N, M, L, c_fftw, r2_fftw, FFTW_ESTIMATE);
    
        fftw_execute(plan_fftw_r2c);
        fftw_execute(plan_fftw_c2r);
        fftw_destroy_plan(plan_fftw_r2c);
        fftw_destroy_plan(plan_fftw_c2r);
    }
    auto stop_0 = std::chrono::high_resolution_clock::now();


    if(use_fftw == 'y')
    {
        std::transform(data_r_2.cbegin(), data_r_2.cend(), data_r_2.begin(), [&N, &M, &L]( T c) { return c/(N*M*L); });
    }

    
    std::cout << "done." << std::endl;
    //CUDA part
    std::cout << "executing cufft...";
    std::cout << std::flush;

    TRealCufft *data_r_1_dev, *data_r_2_dev;
    TComplexCufft* data_c_dev;
    std::size_t size_real = N*M*L;
    std::size_t size_complex = N*M*L_reduced;

    CUDA_SAFE_CALL( hipMallocManaged((void**)&data_r_1_dev, sizeof(TRealCufft)*size_real ) );
    CUDA_SAFE_CALL( hipMallocManaged((void**)&data_r_2_dev, sizeof(TRealCufft)*size_real ) );
    CUDA_SAFE_CALL( hipMallocManaged((void**)&data_c_dev, sizeof(TComplexCufft)*size_complex ) );

    CUDA_SAFE_CALL( hipMemcpy ( data_r_1_dev, data_r_1.data(), sizeof(TRealCufft)*size_real, hipMemcpyHostToDevice ) );
    CUDA_SAFE_CALL( hipMemcpy ( data_r_2_dev, data_r_2.data(), sizeof(TRealCufft)*size_real, hipMemcpyHostToDevice ) );


    auto data_c_dev_c = data_c_dev;
    auto data_r_1_dev_c = data_r_1_dev;
    auto data_r_2_dev_c = data_r_2_dev;
    

    hipfftHandle cufft_handle_r2c, cufft_handle_c2r;

    hipEvent_t start_1, stop_1;
    CUDA_SAFE_CALL( hipEventCreate(&start_1) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_1) );    

    CUDA_SAFE_CALL( hipEventRecord(start_1) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUFFT_SAFE_CALL( hipfftPlan3d(&cufft_handle_r2c, N, M, L, HIPFFT_D2Z) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUFFT_SAFE_CALL( hipfftPlan3d(&cufft_handle_c2r, N, M, L, HIPFFT_Z2D) ); 
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUFFT_SAFE_CALL( hipfftExecD2Z(cufft_handle_r2c, data_r_1_dev_c, data_c_dev_c ) ); //only works with C-style cast!
    CUFFT_SAFE_CALL( hipfftExecZ2D(cufft_handle_c2r, data_c_dev_c, data_r_2_dev_c ) ); //only works with C-style cast!
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    CUFFT_SAFE_CALL( hipfftDestroy(cufft_handle_r2c) );
    CUFFT_SAFE_CALL( hipfftDestroy(cufft_handle_c2r) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUDA_SAFE_CALL( hipEventRecord(stop_1) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_1) );
    
    std::cout << "done." << std::endl;

    float duration_1 = 0;
    CUDA_SAFE_CALL( hipEventElapsedTime(&duration_1, start_1, stop_1) );
    auto duration_0 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_0 - start_0);
    std::cout << " fftw time = " << duration_0.count() << " cufft time = " << duration_1 << std::endl;
    



    hipFree(data_r_1_dev);
    hipFree(data_r_2_dev);
    hipFree(data_c_dev);



    return 0;

}


